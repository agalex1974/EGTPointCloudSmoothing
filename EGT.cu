#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <algorithm>

__device__
void normalize(float& x, float& y, float& z){
    float norm = sqrtf(x * x + y * y + z * z);
    x /= norm; y /= norm; z /= norm;
}

__device__
void GetNormalizedPerpendicularVectorToVector(const float& x1, const float& y1, const float& z1,
                                              float& x2, float& y2, float& z2){
    float max = fabs(x1);

    int cordIndex = 0;
    if (max < fabs(y1))
    {
        cordIndex = 1;
        max = fabs(y1);
    }

    if (max < fabs(z1))
    {
        cordIndex = 2;
    }

    x2 = 1.0;
    y2 = 1.0;
    z2 = 1.0;

    switch (cordIndex)
    {
        case 0:
            x2 = (-y1 * y2 - z1 * z2) / x1;
            break;
        case 1:
            y2 = (-x1 * x2 - z1 * z2) / y1;
            break;
        case 2:
            z2 = (-x1 * x2 - y1 * y2) / z1;
            break;
    }
    normalize(x2, y2, z2);
}

__device__
float norm2(const float& x1, const float& y1, const float& z1, const float& x2, const float& y2, const float& z2){
    return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2);
}
__device__
void cross(const float& u1, const float& u2, const float& u3,
           const float& v1, const float& v2, const float& v3,
           float& x, float&y, float& z){
    x = u2 * v3 - v2 * u3;
    y = v1 * u3 - u1 * v3;
    z = u1 * v2 - v1 * u2;
}

__device__
void matrix_multiplication(float& a11, float& a12, float& a13, float& a14,
                           float& a21, float& a22, float& a23, float& a24,
                           float& a31, float& a32, float& a33, float& a34,
                           float& x, float& y, float& z){
    float x1 = x; float y1 = y; float z1 = z;
    x = a11 * x1 + a12 * y1 + a13 * z1 + a14;
    y = a21 * x1 + a22 * y1 + a23 * z1 + a24;
    z = a31 * x1 + a32 * y1 + a33 * z1 + a34;
}

__device__
void CreateLocalCoordinateSystem(float& xo, float& yo, float& zo,
                                 const float& xd, const float& yd, const float& zd,
                                 float& a11, float& a12, float& a13, float& a14,
                                 float& a21, float& a22, float& a23, float& a24,
                                 float& a31, float& a32, float& a33, float& a34)
{
    GetNormalizedPerpendicularVectorToVector(xd, yd, zd, a21, a22, a23);
    cross(xd, yd, zd, a21, a22, a23, a31, a32, a33);
    normalize(a31, a32, a33);
    a14 = 0.0;
    a24 = 0.0;
    a34 = 0.0;
    matrix_multiplication(a11, a12, a13, a14, a21, a22, a23, a24, a31, a32, a33, a34, xo, yo, zo);
    a14 = -xo;
    a24 = -yo;
    a34 = -zo;
}

__device__
bool isEllipicGabrielNeighbor(int pnt_idx, int i, float* pnts_x, float* pnts_y, float* pnts_z,
                              const float& x, const float& y, const float& z, int* NNs,
                              int count, float a)
{
    int neigh = NNs[i * count + pnt_idx];
    float xi = pnts_x[neigh]; float yi = pnts_y[neigh]; float zi = pnts_z[neigh];
    float xo = 0.5f * (xi + x); float yo = 0.5f * (yi + y); float zo = 0.5f * (zi + z);

    float d = sqrtf(norm2(xi, yi, zi, x, y, z)) / 2.0f;
    float xaxis_x = xi - x; float xaxis_y = yi - y; float xaxis_z = zi - z;
    normalize(xaxis_x, xaxis_y, xaxis_z);
    float a11, a12, a13, a14;
    float a21, a22, a23, a24;
    float a31, a32, a33, a34;
    a11 = xaxis_x; a12 = xaxis_y; a13 = xaxis_z;
    CreateLocalCoordinateSystem(xo, yo, zo, xaxis_x, xaxis_y, xaxis_z, a11, a12, a13, a14,
                                a21, a22, a23, a24, a31, a32, a33, a34);
    for (int j = 0; j < i; j++)
    {
        neigh = NNs[j * count + pnt_idx];
        xi = pnts_x[neigh]; yi = pnts_y[neigh]; zi = pnts_z[neigh];
        matrix_multiplication(a11, a12, a13, a14, a21, a22, a23, a24, a31, a32, a33, a34, xi, yi, zi);
        float ellipsoidValue = xi * xi + yi * yi / (a * a) + zi * zi / (a * a);
        if (ellipsoidValue < d * d) return false;
    }
    return true;
}

__global__
void calculateEGG(float* pnts_x, float* pnts_y, float* pnts_z, int* NNs, float ratio, int count, int neighborsCount, int start, int batchCount)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < batchCount){
        int i = t + start;
        float x = pnts_x[i], y = pnts_y[i], z = pnts_z[i];
        for (int j = neighborsCount - 1; j >= 0; j--)
        {
            if (!isEllipicGabrielNeighbor(i, j, pnts_x, pnts_y, pnts_z, x, y, z, NNs, count, ratio))
                NNs[j * count + i] = -1;
        }
    }
}

__global__
void taubin_step(float* in_x, float* in_y, float* in_z,
                 float* out_x, float* out_y, float* out_z, int count, float scale, int* neighbors, int max_neighbors, int isRegularized, int start, int batchCount){
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < batchCount){
        int i = t + start;
        float cog_x = 0.0f;
        float cog_y = 0.0f;
        float cog_z = 0.0f;
        float sum = 0.0f;
        float x1 = in_x[i]; float y1 = in_y[i]; float z1 = in_z[i];
        for (int n = 0; n < max_neighbors; n++){
            int neigh = neighbors[n * count + i];
            if (neigh != -1) {
                float x2 = in_x[neigh];
                float y2 = in_y[neigh];
                float z2 = in_z[neigh];
                float distance = norm2(x1, y1, z1, x2, y2, z2);
                float w;
                if (isRegularized) w = scale < 0.0f ? 1.0f / (distance + 1e-8f) : 1.0f;
                else w = expf(-distance);
                cog_x += w * (x2 - x1);
                cog_y += w * (y2 - y1);
                cog_z += w * (z2 - z1);
                sum += w;
            }
        }
        out_x[i] = x1 + scale * cog_x / sum;
        out_y[i] = y1 + scale * cog_y / sum;
        out_z[i] = z1 + scale * cog_z / sum;
    }
}

__global__
void scale_points_to_unity(float* x, float* y, float* z, float min, float max, int pointCount)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < pointCount) {
        float d = max - min;
        x[i] = (x[i] - min) / d;
        y[i] = (y[i] - min) / d;
        z[i] = (z[i] - min) / d;
    }
}

__global__
void produce_output(float* x, float* y, float* z, float* xout, float* yout, float* zout, float min, float max, int pointCount)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < pointCount) {
        float d = max - min;
        xout[i] = x[i] * d + min;
        yout[i] = y[i] * d + min;
        zout[i] = z[i] * d + min;
    }
}

void EGTsmoothing(float* in_x, float* in_y, float* in_z, int count, float lambda, float mu,
                  int* neighbors, int maxNeighbors, float* out_x, float* out_y, float* out_z,
                  int iterationCount, int isRegularized, float ratio) {
    size_t size = count * sizeof(float);
	size_t sizeNeighbors = count * maxNeighbors * sizeof(int);
    float *din_x, *din_y, *din_z;
    float *dout_x, *dout_y, *dout_z;
    int *dneighbors;
    hipMalloc((void **) &din_x, size);
    hipMemcpy(din_x, in_x, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &dout_x, size);
    hipMalloc((void **) &din_y, size);
    hipMemcpy(din_y, in_y, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &dout_y, size);
    hipMalloc((void **) &din_z, size);
    hipMemcpy(din_z, in_z, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &dout_z, size);
    hipMalloc((void **) &dneighbors, sizeNeighbors);
    hipMemcpy(dneighbors, neighbors, sizeNeighbors, hipMemcpyHostToDevice);
    int max_threads = 2000000;

    float xmin = thrust::reduce(thrust::device, din_x, din_x + count, 1e8f, thrust::minimum<float>());
    float xmax = thrust::reduce(thrust::device, din_x, din_x + count, -1e8f, thrust::maximum<float>());
    float min = xmin;
    float max = xmax;
    float ymin = thrust::reduce(thrust::device, din_y, din_y + count, 1e8f, thrust::minimum<float>());
    float ymax = thrust::reduce(thrust::device, din_y, din_y + count, -1e8f, thrust::maximum<float>());
    min = std::min(min, ymin);
    max = std::max(max, ymax);
    float zmin = thrust::reduce(thrust::device, din_z, din_z + count, 1e8f, thrust::minimum<float>());
    float zmax = thrust::reduce(thrust::device, din_z, din_z + count, -1e8f, thrust::maximum<float>());
    min = std::min(min, zmin);
    max = std::max(max, zmax);
    int threads = 1024;
    int blocks = (int)ceil((1.0 * count) / threads);
    scale_points_to_unity <<< blocks, threads >> > (din_x, din_y, din_z, min, max, count);
    hipDeviceSynchronize();

	int times = count / max_threads;
    int start = 0;
    for (int i = 0; i < times; i++) {
        int threads = 1024;
        int blocks = (int)ceil((1.0 * max_threads) / threads);
        calculateEGG<<<blocks, threads>>>(din_x, din_y, din_z, dneighbors, ratio, count, maxNeighbors, start, max_threads);
        hipDeviceSynchronize();
        start += max_threads;
    }
    if (start < count){
        int threads = 1024;
        int blocks = (int)ceil((1.0 * (count - start)) / threads);
        calculateEGG<<<blocks, threads>>>(din_x, din_y, din_z, dneighbors, ratio, count, maxNeighbors, start, count - start);
        hipDeviceSynchronize();
    }

    for (int k = 0; k < iterationCount; k++) {
        start = 0;
        for (int i = 0; i < times; i++) {
            int threads = 1024;
            int blocks = (int)ceil((1.0 * (max_threads)) / threads);
            taubin_step<<<blocks, threads>>>(din_x, din_y, din_z,
                                             dout_x, dout_y, dout_z, count, lambda, dneighbors,
                                             maxNeighbors, isRegularized, start, max_threads);
            hipDeviceSynchronize();
            start += max_threads;
        }
        if (start < count){
            int threads = 1024;
            int blocks = (int)ceil((1.0 * (count - start)) / threads);
            taubin_step<<<blocks, threads>>>(din_x, din_y, din_z,
                                             dout_x, dout_y, dout_z, count, lambda, dneighbors,
                                             maxNeighbors, isRegularized, start, count - start);
            hipDeviceSynchronize();
        }
        std::swap(din_x, dout_x);
        std::swap(din_y, dout_y);
        std::swap(din_z, dout_z);

        start = 0;
        for (int i = 0; i < times; i++) {
            int threads = 1024;
            int blocks = (int)ceil((1.0 * (max_threads)) / threads);
            taubin_step<<<blocks, threads>>>(din_x, din_y, din_z,
                                             dout_x, dout_y, dout_z, count, mu, dneighbors,
                                             maxNeighbors, isRegularized, start, max_threads);
            hipDeviceSynchronize();
            start += max_threads;
        }
        if (start < count){
            int threads = 1024;
            int blocks = (int)ceil((1.0 * (count - start)) / threads);
            taubin_step<<<blocks, threads>>>(din_x, din_y, din_z,
                                             dout_x, dout_y, dout_z, count, mu, dneighbors,
                                             maxNeighbors, isRegularized, start, count - start);
            hipDeviceSynchronize();
        }
        std::swap(din_x, dout_x);
        std::swap(din_y, dout_y);
        std::swap(din_z, dout_z);
    }

    threads = 1024;
	blocks = (int)ceil((1.0 * count) / threads);
    produce_output << <blocks, threads >> > (din_x, din_y, din_z, dout_x, dout_y, dout_z, min, max, count);
    hipDeviceSynchronize();

    hipMemcpy(out_x, dout_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(out_y, dout_y, size, hipMemcpyDeviceToHost);
    hipMemcpy(out_z, dout_z, size, hipMemcpyDeviceToHost);
    hipFree(din_x); hipFree(din_y); hipFree(din_z);
    hipFree(dout_x); hipFree(dout_y); hipFree(dout_z);
    hipFree(dneighbors);
}
